
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-1
 * Description: Computation of a matrix C = Kronecker_prod(A, B.T)
 *              where A and B are matrices of dimension (m, n) and
 *              the output is of the dimension (m * n, m * n). 
 * Note: All lines marked in --> should be replaced with code. 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; // The handle for printing the output

__global__ void per_row_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    long int id1=blockIdx.x;
    long int id2=threadIdx.x;

    for(long int j=0 ; j<n ; j++)
    {
      for(long int i=0 ; i<n ; i++)
      {
        C[((id1)*n+i)*m*n+id2+j*m]=A[id1*n+j]*B[id2*n+i];
      }
    }
}

__global__ void per_column_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){    
    long int id1=blockIdx.x;
    long int id2=threadIdx.x;
    long int id3=threadIdx.y;
    long int id=id1*blockDim.x*blockDim.y+id2*blockDim.y+id3;

    if(id<n*n)
    {
      long int id1=id/n;
      long int id2=id%n;

      for(int i=0 ; i<m ; i++)
      {
        for(int j=0 ; j<m ; j++)
        {
          C[(i*n+id2)*m*n+j+id1*m]=A[i*n+id1]*B[j*n+id2];
        }
      }
    }
}

__global__ void per_element_kernel(long int *A, long int *B, long int *C,long int m, long int n){    
    long int id1=blockIdx.x;
    long int id2=blockIdx.y;
    long int id3=threadIdx.x;
    long int id4=threadIdx.y;

    long int ele=(id1*gridDim.y+id2)*(blockDim.x*blockDim.y)+(id3*blockDim.y+id4);

    if(ele<m*m*n*n)
    {
      long int x=ele/(m*n),y=ele%(m*n);

      long int i=x/n,j=y/m,k=y%m,l=x%n;

      C[ele]=A[i*n+j]*B[k*n+l];
    }
}

/**
 * Prints any 1D array in the form of a matrix
 **/
void printMatrix(long int *arr, long int rows, long int cols, char* filename){
    outfile.open(filename);
    for(long int i = 0; i < rows; i++){
        for(long int j = 0; j < cols; j++){
            outfile<<arr[i * cols + j]<<" ";
        }
        outfile<<"\n";
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    long int m,n;	
    cin>>m>>n;	

    // Host_arrays 
    long int *h_a,*h_b,*h_c;

    // Device arrays 
    long int *d_a,*d_b,*d_c;
	
    // Allocating space for the host_arrays 
    h_a = (long int *) malloc(m * n * sizeof(long int));
    h_b = (long int *) malloc(m * n * sizeof(long int));	
    h_c = (long int *) malloc(m * m * n * n * sizeof(long int));	

    // Allocating memory for the device arrays 
    hipMalloc(&d_a,m*n*sizeof(long int));
    hipMalloc(&d_b,m*n*sizeof(long int));
    hipMalloc(&d_c,m*m*n*n*sizeof(long int)); 

    // Read the input matrix A
    for(long int i = 0; i < m * n; i++) {
        cin>>h_a[i];
    }

    //Read the input matrix B 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_b[i];
    }

    // Transfer the input host arrays to the device 
    hipMemcpy(d_a,h_a,n*m*sizeof(long int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,n*m*sizeof(long int),hipMemcpyHostToDevice);

    long int gridDimx, gridDimy;
    
    // Launch the kernels
    /**
     * Kernel 1 - per_row_AB_kernel
     * To be launched with 1D grid, 1D block
     * Each thread should process a complete row of A, B
     **/

    dim3 grid1(m,1,1);
    dim3 block1(m,1,1);

    double starttime = rtclock();  

    per_row_AB_kernel <<<grid1,block1>>> (d_a,d_b,d_c,m,n); 
    hipDeviceSynchronize();                                                           

    double endtime = rtclock(); 
	printtime("GPU Kernel-1 time: ", starttime, endtime);  

    hipMemcpy(h_c,d_c,n*n*m*m*sizeof(long int),hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel1.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(int));

    /**
     * Kernel 2 - per_column_AB_kernel
     * To be launched with 1D grid, 2D block
     * Each thread should process a complete column of  A, B
     **/
    
    gridDimx = ceil(float(n * n) / 1024);
    dim3 grid2(gridDimx,1,1);
    dim3 block2(32,32,1);

    starttime = rtclock(); 

    per_column_AB_kernel <<<grid2,block2>>> (d_a,d_b,d_c,m,n);
    hipDeviceSynchronize(); 

    endtime = rtclock(); 
  	printtime("GPU Kernel-2 time: ", starttime, endtime);  

    hipMemcpy(h_c,d_c,m*m*n*n*sizeof(long int),hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel2.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(int));

    /**
     * Kernel 3 - per_element_kernel
     * To be launched with 2D grid, 2D block
     * Each thread should process one element of the output 
     **/
    gridDimx = ceil(float(n * n) / 16);
    gridDimy = ceil(float(m * m) / 64);
    dim3 grid3(gridDimx,gridDimy,1);
    dim3 block3(64,16,1);

    starttime = rtclock();  

    per_element_kernel <<<grid3,block3>>> (d_a,d_b,d_c,m,n);
    hipDeviceSynchronize();                                                              

    endtime = rtclock();  
	printtime("GPU Kernel-3 time: ", starttime, endtime);  

    hipMemcpy(h_c,d_c,m*m*n*n*sizeof(long int),hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel3.txt");

    return 0;
}
